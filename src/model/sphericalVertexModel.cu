#include "hip/hip_runtime.h"
#include "sphericalVertexModel.cuh"
/*!
    \addtogroup modelKernels
    @{
*/
__global__ void gpu_move_particles_on_sphere_kernel(dVec *pos,
                      dVec *disp,
                      sphericalDomain sphere,
                      scalar scale,
                      int N
                      )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    sphere.move(pos[idx],scale * disp[idx]);
    };

bool gpu_move_particles_on_sphere(dVec *pos,
                                  dVec *disp,
                                  sphericalDomain &sphere,
                                  scalar scale,
                                  int N
                                  )
    {
    unsigned int block_size = 512;
    if (N < 512) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;
    gpu_move_particles_on_sphere_kernel<<<nblocks,block_size>>>(pos,disp,sphere,scale,N);

    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

bool gpu_quadratic_spherical_cellular_force(dVec *cellPos,
                                            dVec *vertexPos,
                                            dVec *forces,
                                            int *vertexCellNeighbors,
                                            unsigned int *vertexCellNeighborNumber,
                                            dVec *currentVertexAroundCell,
                                            dVec *lastVertexAroundCell,
                                            dVec *nextVertexAroundCell,
                                            unsigned int *cellNumberOfNeighbors,
                                            scalar2 *areaPerimeter,
                                            scalar2 *areaPerimeterPreference,
                                            Index2D neighborIndex,
                                            scalar Kr,
                                            int N)
    {
    unsigned int block_size = 512;
    if (N < 512) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;
    //gpu_move_particles_on_sphere_kernel<<<nblocks,block_size>>>(pos,disp,sphere,scale,N);
    UNWRITTENCODE("Force ON GPU");

    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

bool gpu_spherical_vertex_model_geometry(dVec *vertexPos,
                                         dVec *cellPos,
                                         int *cellNeighbors,
                                         int *vertexCellNeighbors,
                                         unsigned int *vertexCellNumberOfNeighbors,
                                         dVec *currentVertexAroundCell,
                                         dVec *lastVertexAroundCell,
                                         dVec *nextVertexAroundCell,
                                         unsigned int *cellNumberOfNeighbors,
                                         scalar2 *areaPerimeter,
                                         Index2D cellNeighborIndex,
                                         Index2D neighborIndex,
                                         int nCells
                                         )
    {
    unsigned int block_size = 512;
    if (nCells < 512) block_size = 32;
    unsigned int nblocks  = nCells/block_size + 1;
    //gpu_move_particles_on_sphere_kernel<<<nblocks,block_size>>>(pos,disp,sphere,scale,N);
    UNWRITTENCODE("GEOMETRY ON GPU");

    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };
/** @} */ //end of group declaration
