#include "hip/hip_runtime.h"
#include "sphericalVertexModel.cuh"
/*!
    \addtogroup modelKernels
    @{
*/
__global__ void gpu_move_particles_on_sphere_kernel(dVec *pos,
                      dVec *disp,
                      sphericalDomain sphere,
                      scalar scale,
                      int N
                      )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    sphere.move(pos[idx],scale * disp[idx]);
    };

bool gpu_move_particles_on_sphere(dVec *pos,
                                  dVec *disp,
                                  sphericalDomain &sphere,
                                  scalar scale,
                                  int N
                                  )
    {
    unsigned int block_size = 512;
    if (N < 512) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;
    gpu_move_particles_on_sphere_kernel<<<nblocks,block_size>>>(pos,disp,sphere,scale,N);

    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

__global__ void gpu_spherical_vertex_model_geometry_kernel(dVec *vertexPos,
                                         dVec *cellPos,
                                         int *cellNeighbors,
                                         int *vertexCellNeighbors,
                                         unsigned int *vertexCellNumberOfNeighbors,
                                         dVec *currentVertexAroundCell,
                                         dVec *lastVertexAroundCell,
                                         dVec *nextVertexAroundCell,
                                         unsigned int *cellNumberOfNeighbors,
                                         scalar2 *areaPerimeter,
                                         Index2D cellNeighborIndex,
                                         Index2D neighborIndex,
                                         sphericalDomain sphere,
                                         int nCells
                                         )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= nCells)
        return;

    int neighs = cellNumberOfNeighbors[idx];
    dVec cPos(0.0);
    for (int nn = 0; nn < neighs;++nn)
        cPos = cPos + vertexPos[cellNeighbors[cellNeighborIndex(nn,idx)]];
    sphere.putInBoxReal(cPos);
    cellPos[idx] = cPos;
        
    int lastVertexIdx = cellNeighbors[cellNeighborIndex(neighs-2,idx)];
    int curVertexIdx = cellNeighbors[cellNeighborIndex(neighs-1,idx)];
    int nextVertexIdx;
    dVec lastVertexPos = vertexPos[lastVertexIdx];
    dVec curVertexPos = vertexPos[curVertexIdx];
    dVec nextVertexPos;
    scalar perimeter = 0.; 
    scalar area = 0.;
    scalar tempVal;
    for (int nn = 0; nn < neighs; ++nn)
        {
        int cni = cellNeighborIndex(nn,idx);
        int vNeighs = vertexCellNumberOfNeighbors[curVertexIdx];
        int forceSetIdx = -1;
        for (int vn = 0; vn < vNeighs; ++vn)
            {
            int newIdx = neighborIndex(vn,curVertexIdx);
            if(vertexCellNeighbors[newIdx] == idx)
                forceSetIdx = newIdx;
            }

        nextVertexIdx = cellNeighbors[cni];
        nextVertexPos = vertexPos[nextVertexIdx];

        sphere.geodesicDistance(lastVertexPos,curVertexPos,tempVal);
        perimeter += tempVal;
        sphere.includedAngle(lastVertexPos,curVertexPos,nextVertexPos,tempVal);
        area += tempVal;

        lastVertexAroundCell[forceSetIdx] = lastVertexPos;
        currentVertexAroundCell[forceSetIdx] = curVertexPos;
        nextVertexAroundCell[forceSetIdx] = nextVertexPos;
        
        lastVertexPos = curVertexPos;
        curVertexIdx = nextVertexIdx;
        curVertexPos = nextVertexPos;
        }
    area = (area-(neighs-2)*PI);
    int extraAngularArea = floor(area/(1.0*PI));
    if(extraAngularArea > 0)
        area -= extraAngularArea*PI;
    area *= (sphere.radius*sphere.radius);

    areaPerimeter[idx].x = area;
    areaPerimeter[idx].y = perimeter;
    };
        
bool gpu_spherical_vertex_model_geometry(dVec *vertexPos,
                                         dVec *cellPos,
                                         int *cellNeighbors,
                                         int *vertexCellNeighbors,
                                         unsigned int *vertexCellNumberOfNeighbors,
                                         dVec *currentVertexAroundCell,
                                         dVec *lastVertexAroundCell,
                                         dVec *nextVertexAroundCell,
                                         unsigned int *cellNumberOfNeighbors,
                                         scalar2 *areaPerimeter,
                                         Index2D cellNeighborIndex,
                                         Index2D neighborIndex,
                                         sphericalDomain &sphere,
                                         int nCells
                                         )
    {
    unsigned int block_size = 512;
    if (nCells < 512) block_size = 32;
    unsigned int nblocks  = nCells/block_size + 1;
    gpu_spherical_vertex_model_geometry_kernel<<<nblocks,block_size>>>
        (vertexPos,cellPos,cellNeighbors,vertexCellNeighbors,vertexCellNumberOfNeighbors,
            currentVertexAroundCell,lastVertexAroundCell,nextVertexAroundCell,
            cellNumberOfNeighbors,areaPerimeter,cellNeighborIndex,neighborIndex,sphere,
            nCells);

    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

__global__ void  gpu_quadratic_spherical_cellular_force_kernel(dVec *cellPos,
                                            dVec *vertexPos,
                                            dVec *forces,
                                            int *vertexCellNeighbors,
                                            unsigned int *vertexCellNeighborNumber,
                                            dVec *currentVertexAroundCell,
                                            dVec *lastVertexAroundCell,
                                            dVec *nextVertexAroundCell,
                                            unsigned int *cellNumberOfNeighbors,
                                            scalar2 *areaPerimeter,
                                            scalar2 *areaPerimeterPreference,
                                            Index2D neighborIndex,
                                            scalar Kr,
                                            sphericalDomain sphere,
                                            int N)
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    dVec vLast,vCur,vNext,cPos,tempVar;
    dVec f(0.0);
    int vNeighs = vertexCellNeighborNumber[idx];
    for (int cc = 0; cc < vNeighs; ++cc)
        {
        dVec fSet(0.0);
        int vni = neighborIndex(cc,idx);
        int cellIndex = vertexCellNeighbors[vni];
        cPos = cellPos[cellIndex];
        vLast = lastVertexAroundCell[vni];
        vCur = currentVertexAroundCell[vni];
        vNext =nextVertexAroundCell[vni];

        scalar areaDifference = areaPerimeter[cellIndex].x - areaPerimeterPreference[cellIndex].x;
        scalar perimeterDifference = areaPerimeter[cellIndex].y - areaPerimeterPreference[cellIndex].y;
            
        dVec thetaHat, phiHat;
        scalar r0, t0, p0;
        sphere.getAngularCoordinates(vCur,r0,t0,p0);
        sphere.cartesianSphericalBasisChange(t0,p0,thetaHat,phiHat);

        sphere.gradientGeodesicDistance(vCur,vLast,tempVar,thetaHat,phiHat);
        fSet -= 2.0*Kr*perimeterDifference*tempVar;
        sphere.gradientGeodesicDistance(vCur,vNext,tempVar,thetaHat,phiHat);
        fSet -= 2.0*Kr*perimeterDifference*tempVar;

        sphere.gradientTriangleArea(vCur,vLast,cPos,tempVar,thetaHat,phiHat);
        fSet -= 2.0*areaDifference*tempVar;
        sphere.gradientTriangleArea(vCur,cPos,vNext,tempVar,thetaHat,phiHat);
        fSet -= 2.0*areaDifference*tempVar;
            
        if(!isnan(fSet[0]))
            f += fSet;
        };

    forces[idx] = f;
    };


bool gpu_quadratic_spherical_cellular_force(dVec *cellPos,
                                            dVec *vertexPos,
                                            dVec *forces,
                                            int *vertexCellNeighbors,
                                            unsigned int *vertexCellNeighborNumber,
                                            dVec *currentVertexAroundCell,
                                            dVec *lastVertexAroundCell,
                                            dVec *nextVertexAroundCell,
                                            unsigned int *cellNumberOfNeighbors,
                                            scalar2 *areaPerimeter,
                                            scalar2 *areaPerimeterPreference,
                                            Index2D neighborIndex,
                                            scalar Kr,
                                            sphericalDomain &sphere,
                                            int N)
    {
    unsigned int block_size = 512;
    if (N < 512) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;
    gpu_quadratic_spherical_cellular_force_kernel<<<nblocks,block_size>>>(cellPos,vertexPos,forces,
                vertexCellNeighbors,vertexCellNeighborNumber,currentVertexAroundCell,lastVertexAroundCell,nextVertexAroundCell,
                cellNumberOfNeighbors,areaPerimeter,areaPerimeterPreference,neighborIndex,Kr,sphere,N);

    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

__global__ void vm_simple_T1_test_kernel(dVec *d_vertexPositions,
                int *d_vertexNeighbors,
                int *d_vertexEdgeFlips,
                int      *d_vertexCellNeighbors,
                unsigned int      *d_cellVertexNum,
                int      *d_cellVertices,
                sphericalDomain &sphere,
                scalar  T1THRESHOLD,
                int      NvTimes3,
                int      vertexMax,
                int      *d_grow,
                Index2D  &cellNeighborIndex)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= NvTimes3)
        return;
    int vertex1 = idx/3;
    int vertex2 = d_vertexNeighbors[idx];
    scalar arcLength;
    if(vertex1 < vertex2)
        {
        sphere.geodesicDistance(d_vertexPositions[vertex1],d_vertexPositions[vertex2],arcLength);
        if(norm(edge) < T1THRESHOLD)
            {
            d_vertexEdgeFlips[idx]=1;

            //test the number of neighbors of the cells connected to v1 and v2 to see if the
            //cell list should grow. This is kind of slow, and I wish I could optimize it away,
            //or at least not test for it during every time step. The latter seems pretty doable.
            //But this is boring, so we'll revisit if optimizations require it
            if(d_cellVertexNum[d_vertexCellNeighbors[3*vertex1]] == vertexMax)
                d_grow[0] = 1;
            if(d_cellVertexNum[d_vertexCellNeighbors[3*vertex1+1]] == vertexMax)
                d_grow[0] = 1;
            if(d_cellVertexNum[d_vertexCellNeighbors[3*vertex1+2]] == vertexMax)
                d_grow[0] = 1;
            if(d_cellVertexNum[d_vertexCellNeighbors[3*vertex2]] == vertexMax)
                d_grow[0] = 1;
            if(d_cellVertexNum[d_vertexCellNeighbors[3*vertex2+1]] == vertexMax)
                d_grow[0] = 1;
            if(d_cellVertexNum[d_vertexCellNeighbors[3*vertex2+2]] == vertexMax)
                d_grow[0] = 1;
            }
        else
            d_vertexEdgeFlips[idx]=0;
        }
    else
        d_vertexEdgeFlips[idx] = 0;
    };

//!Test every edge for a potential T1 event; see if vertexMax needs to increase
bool gpu_vm_test_edges_for_T1(dVec *d_vertexPositions,
                int *d_vertexNeighbors,
                int *d_vertexEdgeFlips,
                int      *d_vertexCellNeighbors,
                unsigned int      *d_cellVertexNum,
                int      *d_cellVertices,
                sphericalDomain &sphere,
                scalar  T1THRESHOLD,
                int      Nvertices,
                int      vertexMax,
                int      *d_grow,
                Index2D  &cellNeighborIndex)
    {
    unsigned int blockSize = 512;
    int nV3 = Nvertices*3;
    if (nV3 < 512) blockSize = 32;
    unsigned int nBlocks = nV3/blockSize + 1;

    vm_simple_T1_test_kernel<<<nBlocks,blockSize>>>(d_vertexPositions,d_vertexNeighbors,
                                                      d_vertexEdgeFlips,d_vertexCellNeighbors,
                                                      d_cellVertexNum,
                                                      Box,T1THRESHOLD,
                                                      nV3,vertexMax,d_grow,cellNeighborIndex);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

bool gpu_vm_parse_multiple_flips(
                    int      *d_vertexEdgeFlips,
                    int      *d_vertexEdgeFlipsCurrent,
                    int      *d_vertexNeighbors,
                    int      *d_vertexCellNeighbors,
                    unsigned int      *d_cellVertexNum,
                    int      *d_cellVertices,
                    int      *d_finishedFlippingEdges,
                    int      *d_edgeFlips,
                    int4     *d_cellSets,
                    Index2D  &cellNeighborIndex,
                    int      Ncells)
    {
    }

bool gpu_vm_flip_edges(
                    int      *d_vertexEdgeFlipsCurrent,
                    dVec *d_vertexPositions,
                    int      *d_vertexNeighbors,
                    int      *d_vertexCellNeighbors,
                    unsigned int      *d_cellVertexNum,
                    int      *d_cellVertices,
                    int      *d_edgeFlips,
                    int4     *d_cellSets,
                    sphericalDomain   &sphere,
                    Index2D  &cellNeighborIndex,
                    int      Nvertices,
                    int      Ncells)
    {
    };
/** @} */ //end of group declaration
